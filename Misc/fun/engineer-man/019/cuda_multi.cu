
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCKS 65535

__global__ void kernel(int *num1, int *num2, int *result) {
    result[blockIdx.x] = num1[blockIdx.x] + num2[blockIdx.x];
}

int main(void) {
    srand(time(NULL));
    int size = sizeof(int) * BLOCKS;

    // host copies
    int *num1, *num2, *result;

    // allocate space on host
    num1 = (int *) malloc(size);
    num2 = (int *) malloc(size);
    result = (int *) malloc(size);

    // device copies
    int *p_num1, *p_num2, *p_result;

    // allocate space on device
    hipMalloc(&p_num1, size);
    hipMalloc(&p_num2, size);
    hipMalloc(&p_result, size);

    // pick numbers to add
    int i;
    for (i = 0; i < BLOCKS; ++i) {
        num1[i] = rand() % 100;
        num2[i] = rand() % 100;
    }

    // copy to device
    hipMemcpy(p_num1, num1, size, hipMemcpyHostToDevice);
    hipMemcpy(p_num2, num2, size, hipMemcpyHostToDevice);

    // start
    kernel<<<BLOCKS,1>>>(p_num1, p_num2, p_result);

    // copy from device
    hipMemcpy(result, p_result, size, hipMemcpyDeviceToHost);

    // print results
    for (i = 0; i < BLOCKS; ++i) {
        printf("%d\n", result[i]);
    }
}
