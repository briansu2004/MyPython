
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *num1, int *num2, int *result) {
    *result = *num1 + *num2;
}

int main(void) {
    // host copies
    int num1, num2, result;

    // device copies
    int *p_num1, *p_num2, *p_result;

    // allocate space on device
    hipMalloc(&p_num1, sizeof(int));
    hipMalloc(&p_num2, sizeof(int));
    hipMalloc(&p_result, sizeof(int));

    // pick numbers to add
    num1 = 4;
    num2 = 5;

    // copy to device
    hipMemcpy(p_num1, &num1, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(p_num2, &num2, sizeof(int), hipMemcpyHostToDevice);

    // start
    kernel<<<1,1>>>(p_num1, p_num2, p_result);

    // copy from device
    hipMemcpy(&result, p_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n", result);
}
